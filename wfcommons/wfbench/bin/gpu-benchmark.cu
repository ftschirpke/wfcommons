#include <iostream>
#include <iomanip>
#include <chrono>
#include <thread>
#include <hiprand.h>
#include <time.h>
#include <math.h>
#include "kernels.cuh"


int main(int argc, char** argv)
{	

	if (argc != 2) {
		std::cerr << "Usage: " << argv[0] << " <work (# 1M samples)>\n";
		exit(1);
    }

	unsigned int n = 256*256;
	unsigned int m;
	unsigned int work_per_thread;
	unsigned int work;
	int *h_count;
	int *d_count;
	hiprandState *d_state;
	// float pi;

	//checking the user input for work
	try {
    	work = std::stol(argv[1]);
    } catch (std::invalid_argument &e) {
        std::cerr << "Invalid argument: " << e.what() << "\n";
        exit(1);
    }

	//making into M samples
	m = 1000000*work;
	work_per_thread = m/(16*16);
	// allocate memory
	h_count = (int*)malloc(n*sizeof(int));
	hipMalloc((void**)&d_count, n*sizeof(int));
	hipMalloc((void**)&d_state, n*sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));


	// set up timing stuff
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);


	// set kernel
	dim3 gridSize = 16;
	dim3 blockSize = 16;
	setup_kernel<<< gridSize, blockSize>>>(d_state);


	// monte carlo kernel
	monte_carlo_kernel<<<gridSize, blockSize>>>(d_state, d_count, work_per_thread);


	// // copy results back to the host
	// hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);


	// // display results and timings for gpu
	// pi = *h_count*4.0/(n*m);
	// std::cout<<"Approximate pi calculated on GPU is: "<<pi<<" and calculation took "<<gpu_elapsed_time<<std::endl;
	std::cout<<"GPU stress test is over and it took "<<gpu_elapsed_time<<std::endl;

	// delete memory
	free(h_count);
	hipFree(d_count);
	hipFree(d_state);
}

